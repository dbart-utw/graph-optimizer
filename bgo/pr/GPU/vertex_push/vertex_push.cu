#include "vertex_push.hpp"

int pr_vertex_push(GPUPRGraph &g, CArray<float> *pr) {
    pr->init(g.num_nodes());
    int max_iters = 100;
    PageRankGPU(g, max_iters, pr->data, VERTEX_PUSH);
    
    return 0;
}
